
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
using namespace std;

__global__ void add(int*a, int*b, int*c)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i]=a[i]+b[i];
}

__managed__ int vector_a[256], vector_b[256],vector_c[256];


int main()
{
    for(int i = 0; i < 256; i++)
    {
        vector_a[i]=i;
        vector_b[i]=256-i;
    }
    add<<<1,256>>>(vector_a,vector_b,vector_c);
    hipDeviceSynchronize();

    int result_sum =0;
    for(int i = 0; i < 256; i++)
    {
        result_sum+=vector_c[i];
    }

    cout<<"sum="<<result_sum<<endl;

    return 0;
}
